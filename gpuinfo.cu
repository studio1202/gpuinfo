#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("GPU Device %d: %s\n", dev, deviceProp.name);
        printf("    Total GPU Memory: %lu bytes\n", (unsigned long)deviceProp.totalGlobalMem);
        
        size_t freeMemory, totalMemory;
        hipMemGetInfo(&freeMemory, &totalMemory);
        printf("    Available GPU Memory: %lu bytes\n", (unsigned long)freeMemory);
    }

    return 0;
}
